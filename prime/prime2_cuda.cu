//
// 素数計算 (NVIDIA CUDA)
//
// nvcc -std=c++11 prime2_cuda.cu 
// 

#include <hip/hip_runtime.h>
#include <iostream>
#include<stdio.h>
#include<array>
#include<memory>

///#include "make_unique.hpp" /// for c++11
using namespace std;

//const int MAX_INT = 1000000;
///const unsigned int MAX_INT = 1024;
const unsigned int MAX_INT       = 200000;
const unsigned int NUM_OF_THREAD = 1024;

//
// Prime 素数を判定(Devcice)
//
__device__ bool isPrime(unsigned int val) {
  unsigned int  half_val = (val >> 1); /// val / 2 
  bool          ret_code = false;

  unsigned int i;
  for (i = half_val; i>1; i--) {
    if ((val % i) == 0) break;
  }
  if (i == 1) ret_code = true;
  else        ret_code = false;

  return ret_code;
}


//
// カーネル関数(Devcice)
//
__global__ void Kernel_Prime(
 const unsigned int *in,
       unsigned int *rst,
       unsigned int  base   ) {

  unsigned int offset = *(in + base + threadIdx.x) ;

  // 判定
  if( isPrime(*(in+offset)) ){ 
    *(rst+offset) = *(in+offset);
  }else{
    *(rst+offset) = 0;
  }
  
}



//
//
int main(){
  unsigned int  *inVal_h;
  unsigned int  *inVal_d;
  unsigned int  *rstVal_d;    /// 素数判定結果
  unsigned int  *rstVal_h;    /// 素数判定結果6
  
  hipError_t cudaStatus;

  /// GPU上に入力値のバッファを確保(Device)
  hipMalloc((void**)&inVal_d, MAX_INT * sizeof(unsigned int));
  
  ///        判定結果の保存バッファを確保(Device)
  hipMalloc((void**)&rstVal_d, MAX_INT * sizeof(unsigned int));

  /// 入力値を用意(Host)
  inVal_h = new unsigned int[MAX_INT];
  for (unsigned int i = 0; i<MAX_INT; i++) {
    inVal_h[i] = i;
  }
  /// 入力値を、GPU上に転送(Host -> Device) 
  hipMemcpy(inVal_d, inVal_h,
             MAX_INT * sizeof(unsigned int),
             hipMemcpyHostToDevice); /* メモリ転送(Host→Device) */

  ///
  /// CUDA カーネル実行
  ///
  printf("start\n");
  for(unsigned int base=0; base < MAX_INT; base+=NUM_OF_THREAD ){   
    cout << ".." << base << endl;

    Kernel_Prime<<<1,NUM_OF_THREAD>>>(inVal_d, rstVal_d, base);

    //
    // 終了持ち
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
      fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
      ///goto Error;
    }

  }
  
  
  /// 結果を取得(Device -> Host) 
  rstVal_h = new unsigned int[MAX_INT];
  hipMemcpy(rstVal_h, rstVal_d,
             MAX_INT * sizeof(unsigned int),
             hipMemcpyDeviceToHost);  /* メモリ転送(Device→Host) */

  /// 結果を表示
  for (int j = 1; j<MAX_INT; j++) {
    if (rstVal_h[j] != 0) printf("%d ", j);
  }


  
  /// MemoryFree
  delete[] inVal_h;
  delete[] rstVal_h;
  hipFree(inVal_d);
  hipFree(rstVal_d);

  return 0;
}
